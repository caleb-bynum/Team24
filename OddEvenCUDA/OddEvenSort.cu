#include "hip/hip_runtime.h"
/* 
CUDA Implementation Pseudocode

Variables:
    n : number of elements to sorted
    t : number of threads
    A : array to be sorted

Idea: avoid coordination between threads by strategically launching kernels
      and subsequently synchronzing.

Main Procedure:
    1. Copy A to GPU memory
    2. for each phase in {0, 1, ..., n-1}:
    3.   if phase is even:
    4.     launch EvenPhase kernel
    5.   else:
    6.     launch OddPhase kernel
    7.   synchronize device
    8. Copy A to device memory

EvenPhase Kernel:
    1. Id = threadId + blockDim * blockId
    2. index1 = 2 * Id
    3. index2 = index1 + 1
    4. compare and swap array elements at index1, index2

OddPhase Kernel:
    1. Id = threadId + blockDim * blockId
    2. index1 = 2 * Id + 1
    3. index2 = index1 + 1
    4. if thread is not the last thread
    5.   compare and swap array elements at index1, index2
*/


#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>


int THREADS_PER_BLOCK;
int BLOCKS;
int NUM_VALS;

float random_float() {
    return (float)rand() / (float)RAND_MAX;
}

/* random input */
void fill_array_random(float* A, int n) {
    for (int i = 0; i < n; i++) {
        A[i] = random_float();
    }
}

/* sorted input */
void fill_array_sorted(float* A, int n) {
    for (int i = 0; i < n; i++) {
        A[i] = (float)i / (float)n;
    }
}

/* reverse sorted input */
void fill_array_reverse_sorted(float* A, int n) {
    for (int i = 0; i < n; i++) {
        A[i] = (float)(n - i) / (float)n;
    }
}

/* print for debugging purposes */
void print_array(float* A, int n) {
    for (int i = 0; i < n; i++) {
        printf("%f ", A[i]);
    }
    printf("\n");
}

/* check that array is sorted */
bool correctness_check(float* A, int n) {
    for (int i = 0; i < n - 1; i++) {
        if (A[i] > A[i+1]) {
            return false;
        }
    }
    return true;
}

/* compare and swap for CUDA threads */
__device__ void compare_and_swap(float* A, int i, int j) {
    if (A[i] > A[j]) {
        float temp = A[i];
        A[i] = A[j];
        A[j] = temp;
    }
}

__global__ void even_phase(float* A) {
    int Id = threadIdx.x + blockDim.x * blockIdx.x;
    int index1 = 2 * Id;
    int index2 = index1 + 1;

 
    compare_and_swap(A, index1, index2);

}

__global__ void odd_phase(float* A, int numVals) {
    
   
    int Id = threadIdx.x + blockDim.x * blockIdx.x;
    int index1 = 2 * Id + 1;
    int index2 = index1 + 1;
  


    if (index2 < numVals) {
        compare_and_swap(A, index1, index2);
    }
   
}

void odd_even_sort(float* A, int numVals) {
    /* allocate space on GPU */
    float* device_A;
    size_t size_bytes = NUM_VALS * sizeof(float);
    hipMalloc( (void**)&device_A, size_bytes );

    /* copy CPU array to GPU */
    CALI_MARK_BEGIN("comm");
    CALI_MARK_BEGIN("comm_large");
    hipMemcpy( device_A, A, size_bytes, hipMemcpyHostToDevice );
    CALI_MARK_END("comm_large");
    CALI_MARK_END("comm");

    /* iterate through each phase, launching the appropriate kernel */
    for (int phase = 0; phase < NUM_VALS; phase++) {
        if (phase % 2 == 0) {
            CALI_MARK_BEGIN("comp");
            CALI_MARK_BEGIN("comp_large");
            even_phase<<<BLOCKS, THREADS_PER_BLOCK>>>(device_A);
            CALI_MARK_END("comp_large");
            CALI_MARK_END("comp");
        } else {
            CALI_MARK_BEGIN("comp");
            CALI_MARK_BEGIN("comp_large");
            odd_phase<<<BLOCKS, THREADS_PER_BLOCK>>>(device_A, numVals);
            CALI_MARK_END("comp_large");
            CALI_MARK_END("comp");
        }

        CALI_MARK_BEGIN("comm");
        CALI_MARK_BEGIN("comm_small");
        hipDeviceSynchronize();
        CALI_MARK_END("comm_small");
        CALI_MARK_END("comm");
    }

    /* copy GPU array back to Host */
    CALI_MARK_BEGIN("comm");
    CALI_MARK_BEGIN("comm_large");
    hipMemcpy( A, device_A, size_bytes, hipMemcpyDeviceToHost );
    CALI_MARK_END("comm_large");
    CALI_MARK_END("comm");

    /* free GPU memory */
    hipFree(device_A);
}

int main(int argc, char** argv) {
    /* initialize random seed */
    srand(time(NULL));

    /* Initialize CALI */
    cali::ConfigManager mgr;
    mgr.start();
    CALI_MARK_BEGIN("main");

    /* get command line arguments */
    if (argc != 3) {
        printf("Usage: ./odd_even_sort <num_vals> <threads_per_block>\n");
        exit(1);
    }

    NUM_VALS = atoi(argv[1]);
    // print NUM_VALS
    printf("NUM_VALS: %d\n", NUM_VALS);

    THREADS_PER_BLOCK = atoi(argv[2]);
    // print THREADS_PER_BLOCK
    printf("THREADS: %d\n", THREADS_PER_BLOCK);
    BLOCKS = NUM_VALS / THREADS_PER_BLOCK;

    /* Adiak Variables */
    std::string algorithm = "OddEvenSort";
    std::string programmingModel = "CUDA";
    std::string datatype = "float";
    int sizeOfDatatype = sizeof(float);
    int inputSize = NUM_VALS;
    std::string inputType = "Random";
    int num_threads = THREADS_PER_BLOCK;
    int num_blocks = BLOCKS;
    int group_number = 1;
    std::string implementation_source = "Handwritten";


    /* allocate space for array */
    float* A = (float*)malloc(NUM_VALS * sizeof(float));

    /* fill array with random values */
    CALI_MARK_BEGIN("data_init");
    fill_array_random(A, NUM_VALS);
    CALI_MARK_END("data_init");

    /* sort array */
    odd_even_sort(A, NUM_VALS);

    /* print sorted array */
    // print_array(A, NUM_VALS);

    /* check correctness */
    CALI_MARK_BEGIN("correctness_check");
    if (correctness_check(A, NUM_VALS)) {
        printf("Correctness check passed\n");
    } else {
        printf("Correctness check failed\n");
    }
    CALI_MARK_END("correctness_check");

    /* Adiak metadata */
    adiak::init(NULL);
    adiak::launchdate();    // launch date of the job
    adiak::libraries();     // Libraries used
    adiak::cmdline();       // Command line used to launch the job
    adiak::clustername();   // Name of the cluster
    adiak::value("Algorithm", algorithm); // The name of the algorithm you are using (e.g., "MergeSort", "BitonicSort")
    adiak::value("ProgrammingModel", programmingModel); // e.g., "MPI", "CUDA", "MPIwithCUDA"
    adiak::value("Datatype", datatype); // The datatype of input elements (e.g., double, int, float)
    adiak::value("SizeOfDatatype", sizeOfDatatype); // sizeof(datatype) of input elements in bytes (e.g., 1, 2, 4)
    adiak::value("InputSize", inputSize); // The number of elements in input dataset (1000)
    adiak::value("InputType", inputType); // For sorting, this would be "Sorted", "ReverseSorted", "Random", "1%perturbed"
    adiak::value("num_threads", num_threads); // The number of CUDA or OpenMP threads
    adiak::value("num_blocks", num_blocks); // The number of CUDA blocks 
    adiak::value("group_num", group_number); // The number of your group (integer, e.g., 1, 10)
    adiak::value("implementation_source", implementation_source); // Where you got the source code of your algorithm; choices: ("Online", "AI", "Handwritten").
    
    /* finalize program */
    free(A);
    mgr.stop();
    mgr.flush();
    CALI_MARK_END("main");
}
