#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include <hip/hip_runtime.h>

#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>
#include "../helper.h"

int THREADS;
int BLOCKS;
int NUM_VALS;


/* Define Caliper region names */
const char* comm = "comm";
const char* comm_large = "comm_large";
const char* comp = "comp";
const char* comp_large = "comp_large";

int bitonic_counter = 0;


__global__ void enumerationSort(float *array, int *rank, int n, int THREADS) {
    long int k = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i = k; i < n; i += THREADS){
        
        if (i < n) {
            rank[i] = 0;
            for (int j = 0; j < n; j++) {
                if (array[j] < array[i] || (array[j] == array[i] && j < i)) {
                    rank[i]++;
                }
            }
        }
    }
}

// CUDA kernel for sorting the array based on ranks
__global__ void sortArray(float *array, float *sorted_array, int *rank, int n, int THREADS) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = k; i < n; i += THREADS){
        sorted_array[rank[i]] = array[i];
    }
}


int main(int argc, char *argv[])
{
  
    CALI_MARK_BEGIN("main");
    THREADS = atoi(argv[1]);
    NUM_VALS = atoi(argv[2]);
    BLOCKS = NUM_VALS / THREADS;
    std::string input_type = argv[3];


    // Create caliper ConfigManager object
    cali::ConfigManager mgr;
    mgr.start();
    const int n = NUM_VALS; // Size of the array


    CALI_MARK_BEGIN("data_init");
    // printf("cali 0\n");
    float *h_array = new float[n];
    // printf("cali 1\n");
    float *sorted_array = new float[n];

    // printf("cali 2\n");

    // Initialize the array with random values
    array_fill(h_array, n, input_type);
    // printf("cali 3\n");
    CALI_MARK_END("data_init");


    // Device arrays
    float *d_array, *sorted_array_device;
    int* d_rank;
    hipMalloc((void**)&d_array, sizeof(float) * n);
    hipMalloc((void**)&d_rank, sizeof(int) * n);
    hipMalloc((void**)&sorted_array_device, sizeof(float) * n);

    // printf("Cuda arrays allocated\n");

    CALI_MARK_BEGIN("comm");
    CALI_MARK_BEGIN("comm_large");
    CALI_MARK_BEGIN("hipMemcpy");
    // Copy data from host to device
    hipMemcpy(d_array, h_array, sizeof(float) * n, hipMemcpyHostToDevice);
    CALI_MARK_END("hipMemcpy");
    CALI_MARK_END("comm_large");
    CALI_MARK_END("comm");


    CALI_MARK_BEGIN("comp");
    CALI_MARK_BEGIN("comp_large");

    // Launch the enumeration sort kernel
    enumerationSort<<<BLOCKS, THREADS>>>(d_array, d_rank, n, THREADS);
    hipDeviceSynchronize();
    CALI_MARK_END("comp_large");

    CALI_MARK_BEGIN("comp_large");
    // Launch the sorting kernel to rearrange the array
    sortArray<<<BLOCKS, THREADS>>>(d_array, sorted_array_device, d_rank, n, THREADS);
    hipDeviceSynchronize();

    CALI_MARK_END("comp_large");
    CALI_MARK_END("comp");


    CALI_MARK_BEGIN("comm");
    

    CALI_MARK_BEGIN("comm_large");
    CALI_MARK_BEGIN("hipMemcpy");
    hipMemcpy(sorted_array, sorted_array_device, sizeof(float) * n, hipMemcpyDeviceToHost);
    CALI_MARK_END("hipMemcpy");
    CALI_MARK_END("comm_large");

    CALI_MARK_END("comm");


    CALI_MARK_BEGIN("correctness_check");
    bool correct = correctness_check(sorted_array, NUM_VALS);
    CALI_MARK_END("correctness_check");
    if(correct) printf("Array correctly sorted\n");

 
    // Clean up
    delete[] h_array;
    delete[] sorted_array;
    hipFree(d_array);
    hipFree(d_rank);
    hipFree(sorted_array_device);

    CALI_MARK_END("main");


    adiak::init(NULL);
    adiak::launchdate();    // launch date of the job
    adiak::libraries();     // Libraries used
    adiak::cmdline();       // Command line used to launch the job
    adiak::clustername();   // Name of the cluster
    adiak::value("Algorithm", "EnumerationSort"); // The name of the algorithm you are using (e.g., "MergeSort", "BitonicSort")
    adiak::value("ProgrammingModel", "CUDA"); // e.g., "MPI", "CUDA", "MPIwithCUDA"
    adiak::value("Datatype", "float"); // The datatype of input elements (e.g., double, int, float)
    adiak::value("SizeOfDatatype", 4); // sizeof(datatype) of input elements in bytes (e.g., 1, 2, 4)
    adiak::value("InputSize", NUM_VALS); // The number of elements in input dataset (1000)
    adiak::value("InputType", input_type); // For sorting, this would be "Sorted", "ReverseSorted", "Random", "1%perturbed"
    // adiak::value("num_procs", ); // The number of processors (MPI ranks)
    adiak::value("num_threads", THREADS); // The number of CUDA or OpenMP threads
    adiak::value("num_blocks", BLOCKS); // The number of CUDA blocks 
    adiak::value("group_num", 15); // The number of your group (integer, e.g., 1, 10)
    adiak::value("implementation_source", "Handwritten"); // Where you got the source code of your algorithm; choices: ("Online", "AI", "Handwritten").
  

  // Flush Caliper output before finalizing MPI
  mgr.stop();
  mgr.flush();
}